#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10

#ifndef checkCudaErrors
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

void __checkCudaErrors(hipError_t err, const char *file, const int line)
{

  if(hipSuccess != err)
  {
    fprintf(stderr, "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, line %i.\n", err, hipGetErrorString(err), file, line); 
    exit(EXIT_FAILURE);
  }

}

#endif

__global__ void add(int *a, int *b, int *c)
{
  int tid = blockIdx.x;
  if (tid < N)
    c[tid] = a[tid] + b[tid];
}

int main(void)
{

  int a[N], b[N], c[N]; 
  int *dev_a, *dev_b, *dev_c;


  checkCudaErrors(hipMalloc((void**)&dev_a, 10000000000000 * sizeof(int)));

  hipMalloc((void**)&dev_b, N * sizeof(int));
  hipMalloc((void**)&dev_c, N * sizeof(int));

  for (int i = 0; i < N; ++i)
  {
    a[i] = -i; 
    b[i] = i * i; 
  }

  hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

  add<<<N,1>>>(dev_a, dev_b, dev_c);

  hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost); 

  for (int i = 0; i < N; ++i)
  {
    printf("%d + %d = %d\n",a[i],b[i],c[i]); 
  }

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  return 0;
}

