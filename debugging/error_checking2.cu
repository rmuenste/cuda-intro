#include <stdio.h>
#include <hip/hip_runtime.h>

void checkCudaErrors(hipError_t err, const char *userLabel)
{

  if(hipSuccess != err)
  {
    fprintf(stderr, "checkCudaErrors() Driver API error = %04d \"%s\" at user label \"%s\".\n", err, hipGetErrorString(err), userLabel); 
    exit(EXIT_FAILURE);
  }

}

int main(void)
{

  int *dev_a;

  checkCudaErrors(hipMalloc((void**)&dev_a, 10000000000000 * sizeof(int)),"allocating dev_a");

  hipFree(dev_a);

  return 0;
}

