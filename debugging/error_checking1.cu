#include <stdio.h>
#include <hip/hip_runtime.h>

int main(void)
{

  int *dev_a;

  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;

  err = hipMalloc((void**)&dev_a, 10000000000000 * sizeof(int));

  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to allocate device vector, error message: \"%s\"!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  hipFree(dev_a);

  return 0;
}


