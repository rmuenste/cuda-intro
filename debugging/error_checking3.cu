#include <stdio.h>
#include <hip/hip_runtime.h>

#ifndef checkCudaErrors
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

void __checkCudaErrors(hipError_t err, const char *file, const int line)
{

  if(hipSuccess != err)
  {
    fprintf(stderr, "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, line %i.\n", err, hipGetErrorString(err), file, line); 
    exit(EXIT_FAILURE);
  }

}

#endif

int main(void)
{

  int *dev_a;

  checkCudaErrors(hipMalloc((void**)&dev_a, 10000000000000 * sizeof(int)));

  hipFree(dev_a);

  return 0;
}

