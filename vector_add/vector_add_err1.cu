#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10

__global__ void add(int *a, int *b, int *c)
{
  int tid = blockIdx.x;
  if (tid < N)
    c[tid] = a[tid] + b[tid];
}

int main(void)
{

  int a[N], b[N], c[N]; 
  int *dev_a, *dev_b, *dev_c;


  hipMalloc((void**)&dev_a, 10000000000000 * sizeof(int));
  hipMalloc((void**)&dev_b, N * sizeof(int));
  hipMalloc((void**)&dev_c, N * sizeof(int));

  for (int i = 0; i < N; ++i)
  {
    a[i] = -i; 
    b[i] = i * i; 
  }

  hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

  add<<<N,1>>>(dev_a, dev_b, dev_c);

  hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost); 

  for (int i = 0; i < N; ++i)
  {
    printf("%d + %d = %d\n",a[i],b[i],c[i]); 
  }

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  return 0;
}


//  // Error code to check return values for CUDA calls
//  cudaError_t err = cudaSuccess;
//
//  err = cudaMalloc((void**)&dev_a, 10000000000000 * sizeof(int));
//
//  if (err != cudaSuccess)
//  {
//    fprintf(stderr, "Failed to allocate device vector a (error code %s)!\n", cudaGetErrorString(err));
//    exit(EXIT_FAILURE);
//  }
